#include "hip/hip_runtime.h"
#include "car.h"

#include <cmath>
#include <iostream>

typedef Eigen::Vector2f vec2f;

float MEAN_ACC = 40;
float FWHM_ACC = 10;
float MEAN_VEL = 50;
float FWHM_VEL = 10;
float MEAN_STEER = 10;
float FWHM_STEER = 5;
int CAR_WIDTH = 5;
int CAR_LENGTH = 10;

#define CUDA_WARN(XXX) \
    do { if (XXX != hipSuccess) {std::cerr << "CUDA Error: " << \
    hipGetErrorString(XXX) << ", at line " << __LINE__ \
    << " in file " __FILE__ << std::endl; \
    hipDeviceSynchronize(); exit(EXIT_FAILURE);} } while (0)

CarPool::CarPool(int size): size(size) {
    this->index = 0;
    this->acc_dist = std::normal_distribution<float>(MEAN_ACC, FWHM_ACC);
    this->vel_dist = std::normal_distribution<float>(MEAN_VEL, FWHM_VEL);
    this->steer_dist = std::normal_distribution<float>(MEAN_STEER, FWHM_STEER);

    this->host_cars = new Car[size];
    for (int i=0; i<this->size; i++) {
        this->host_cars[i].alive = 0;
        this->host_cars[i].id = -1;
        this->host_cars[i].pos = vec2f::Zero();
        this->host_cars[i].vel = vec2f::Zero();
        this->host_cars[i].acc = vec2f::Zero();
    }

    CUDA_WARN(hipMalloc(&(this->device_cars), size * sizeof(Car)));
    CUDA_WARN(hipMemcpy(this->device_cars, this->host_cars, size * sizeof(Car), hipMemcpyHostToDevice));
}

int CarPool::new_car(vec2f pos) {
    Car new_car;
    new_car.id = this->index;
    new_car.alive = true;
    new_car.pos = pos;
    new_car.vel = vec2f::Zero();
    new_car.acc = vec2f::Zero();

    new_car.max_acc = this->acc_dist(this->random_generator);
    new_car.max_vel = this->vel_dist(this->random_generator);
    new_car.max_steer = this->steer_dist(this->random_generator);

    std::cout << "New car created." << std::endl
        << "\tid: " << new_car.id << std::endl
        << "\tmax acc: " << new_car.max_acc << std::endl
        << "\tmax vel: " << new_car.max_vel << std::endl
        << "\tmax steer: " << new_car.max_steer << std::endl;

    CUDA_WARN(hipMemcpy(this->device_cars + this->index, &new_car, sizeof(Car), hipMemcpyHostToDevice));
    this->index++;

    return new_car.id;
}

__device__ float distance_to_car_in_front(Car& car, Car* cars, int cur_car_index) {
    // Arbitratrily large distance acting as infinity.
    float distance = 1000000;
    for (int i=0; i<cur_car_index; i++) {
        Car other_car = cars[i];
        if (other_car.id == car.id) continue;
        // Other car must be in front of this one.
        if (car.vel.dot(other_car.pos - car.pos) < 0) continue;
        float distance_i = (car.pos - other_car.pos).norm();
        distance = min(distance_i, distance);
    }
    return distance;
}

__device__ vec2f get_perpendicular_clockwise(vec2f vec) {
    return vec2f(vec(1), -vec(0)).normalized();
}

__device__ vec2f steer_towards(Car car, vec2f goal) {
    vec2f perp_clockwise = get_perpendicular_clockwise(car.vel);
    vec2f perp_anticlockwise = -perp_clockwise;
    vec2f acc = vec2f::Zero();

    if (perp_anticlockwise.dot(goal - car.pos) > 0.0001 ) {
        acc = car.max_steer * perp_anticlockwise;
    }
    if (perp_clockwise.dot(goal - car.pos) > 0.0001 ) {
        acc = car.max_steer * perp_clockwise;
    }

    return acc;
}

__device__ float sigmoid(float x) {
    float res = 1 / (1 +  exp(-x))-.5;
    if (res > 0) res = 0;
    return res;
}

__device__ vec2f accelerate_car_towards(Car car, vec2f goal, Car* cars, int cur_car_index) {
    float dist_next_car = distance_to_car_in_front(car, cars, cur_car_index);

    vec2f dir;
    if (car.vel.norm() < car.max_vel * 0.01) {
        dir = (goal - car.pos).normalized();
    } else {
        dir = car.vel.normalized();
    }
    vec2f acc = dir * car.max_acc;

    if (dist_next_car < 0.6 * car.max_vel) {
        float scared_factor = sigmoid(dist_next_car - car.vel.norm());
        acc *= scared_factor;
    }

    return acc;
}

__device__ vec2f find_goal(vec2f pos) {
    return pos + vec2f(300, 0);
}

__global__ void behaviour_kernel(int road_length, Car* cars, int cur_car_index) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i > cur_car_index - 1) return;
    Car car = cars[i];
    if (not car.alive) return;
    if (car.pos(0) > road_length) {
        car.pos -= vec2f(road_length, 0);
    }
    vec2f goal = find_goal(car.pos);
    car.acc = accelerate_car_towards(car, goal, cars, cur_car_index);
    car.acc += steer_towards(car, goal);
    cars[i] = car;
}

void CarPool::behaviour(Road* road) {
    behaviour_kernel<<< 1 + this->size/256, 256>>>(road->length, this->device_cars, this->index);
    hipDeviceSynchronize();
}

__global__ void physics_kernel(float dT, Car* cars, int cur_car_index) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i > cur_car_index) return;
    Car car = cars[i];
    if (!car.alive) return;
    car.pos += car.vel * dT;
    car.vel += car.acc * dT;

    // Implement a physical speed limit.
    if (car.vel.norm() > car.max_vel) {
        car.vel.normalize();
        car.vel *= car.max_vel;
    }
    cars[i] = car;
}

__host__ void CarPool::physics(float dT) {
    physics_kernel<<< 1 + this->size/256, 256>>>(dT, this->device_cars, this->index);
    hipDeviceSynchronize();
}

void CarPool::display(SDL_Manager& sdl, Road* road) {
    CUDA_WARN(hipMemcpy(this->host_cars, this->device_cars, size * sizeof(Car), hipMemcpyDeviceToHost));

    SDL_SetRenderDrawColor(sdl.rend, 255, 0, 0, 255);
    for (int i=0;i<this->index;i++) {
        Car car = this->host_cars[i];
        if (!car.alive) continue;
        vec2f pos = road->get_display_position(car.pos);
        SDL_Rect rect;
        rect.x = pos(0);
        rect.y = pos(1);
        rect.w = CAR_LENGTH;
        rect.h = CAR_WIDTH;
        SDL_RenderFillRect(sdl.rend, &rect);
    }
}

CarPool::~CarPool() {
    hipFree(this->device_cars);
    free(this->host_cars);
}
